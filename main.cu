
#include "cnn/propagation.cu"
#include "mnist_load.c"

#include <unistd.h> //for sleep in test()

static mnist_data *train_set, *test_set;
static unsigned int train_cnt, test_cnt;

// Data and model loading methods
static inline void load_data(int if_train)
{
    if(if_train)
        mnist_load("dataset/train-images-idx3-ubyte", "dataset/train-labels-idx1-ubyte",
            &train_set, &train_cnt);
    else 
        mnist_load("dataset/t10k-images-idx3-ubyte", "dataset/t10k-labels-idx1-ubyte",
            &test_set, &test_cnt);
}

// Unfold the input layer
static void unfold_input(double input[28][28], double unfolded[24*24][5*5])
{
    int a = 0;
    (void)unfold_input;

    for (int i = 0; i < 2; ++i)
        for (int j = 0; j < 2; ++j) {
            int b = 0;
            for (int x = i; x < i + 2; ++x)
                for (int y = j; y < j+2; ++y)
                    unfolded[a][b++] = input[x][y];
            a++;
        }
}

static void learn(int iter)
{
    static hipblasHandle_t blas;
    hipblasCreate(&blas);

    float err;
    
    double time_taken = 0.0;

    fprintf(stdout ,"Learning\n");

    while (iter < 0 || iter-- > 0) {
        err = 0.0f;

        for (int i = 0; i < train_cnt; ++i) {
            float tmp_err;

            time_taken += forward_propagation(train_set[i].data);

            l_f.bp_clear();
            l_s1.bp_clear();
            l_c1.bp_clear();

            // Euclid distance of train_set[i]
            makeError<<<10, 1>>>(l_f.d_preact, l_f.output, train_set[i].label, 10);
            hipblasSnrm2(blas, 10, l_f.d_preact, 1, &tmp_err);
            err += tmp_err;

            time_taken += back_propagation();
        }

        err /= train_cnt;
        fprintf(stdout, "error: %e, time_on_gpu: %lf\n", err, time_taken);

        if (err < threshold) {
            fprintf(stdout, "Training complete, error less than threshold\n\n");
            break;
        }

    }
    
    fprintf(stdout, "\n Time - %lf\n", time_taken);
}

// Perform forward propagation of test data
static void test()
{
    int error = 0, res; char opt;
    fprintf(stdout, "Show images? [y/n]:");
    fscanf(stdin, "%c", &opt);
    for (int i = 0; i < test_cnt; ++i) {
        if(opt == 'y')
            fprintf(stdout, "\033[2J\033[1;1H");
        res = classify(test_set[i].data, opt);
        if (res != test_set[i].label)
            ++error;
        if(opt == 'y') {
            fprintf(stdout, "\033[1;3");
            if (res != test_set[i].label)
                fprintf(stdout, "1");
            else
                fprintf(stdout, "2");
            fprintf(stdout, "m  %f percent correct network\n  ████████████████████████\n  ████████████████████████\033[0m", 100 * ( 1 - error /float(1 + i)));
            sleep(1);
        }
    }
    fprintf(stdout, "%f percent correct network\n", 100 * ( 1 - error /float(test_cnt)));
}

int main(int argc, const  char **argv)
{
    srand(time(NULL));

    hipError_t err = hipInit(0);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA initialisation failed with error code - %d\n", err);
        return 1;
    }
    if (argc == 2) {
        load_data(1);
        load_model();
        learn(atoi(argv[1]));
        save_model();
    } else {
        load_data(0);
        load_model();
        test();
    }

    return 0;
}
