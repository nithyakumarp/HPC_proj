

#include <hip/hip_runtime.h>
#include <cstdio>

class Layer_info {
    public:
    int M, N, O;

    float *opt;
    float *preact;

    float *bias;
    float *weight;

    float *bp_opt;
    float *bp_preact;
    float *bp_weight;

    Layer_info(int M, int N, int O);

    ~Layer_info();

    void setOutput(float *data);
    void clear();
    void bp_clear();
    void save_model(FILE*);
    void read_model(FILE*);
};

// Constructor
Layer_info::Layer_info(int M, int N, int O)
{
    this->M = M;
    this->N = N;
    this->O = O;

    float h_bias[N];
    float h_weight[N][M];

    opt = NULL;
    preact = NULL;
    bias   = NULL;
    weight = NULL;

    for (int i = 0; i < N; ++i) {
        h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);
        /*h_bias[i] = 0.0f;*/

        for (int j = 0; j < M; ++j) {
            h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);
            /*h_weight[i][j] = 0.05f;*/
        }
    }

    hipMalloc(&opt, sizeof(float) * O);
    hipMalloc(&preact, sizeof(float) * O);

    hipMalloc(&bias, sizeof(float) * N);

    hipMalloc(&weight, sizeof(float) * M * N);

    hipMalloc(&bp_opt, sizeof(float) * O);
    hipMalloc(&bp_preact, sizeof(float) * O);
    hipMalloc(&bp_weight, sizeof(float) * M * N);

    hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);

    hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

// Destructor
Layer_info::~Layer_info()
{
    hipFree(opt);
    hipFree(preact);

    hipFree(bias);

    hipFree(weight);

    hipFree(bp_opt);
    hipFree(bp_preact);
    hipFree(bp_weight);
}

// Send data one row from dataset to the GPU
void Layer_info::setOutput(float *data)
{
    hipMemcpy(opt, data, sizeof(float) * O, hipMemcpyHostToDevice);
}

// Reset GPU memory between iterations
void Layer_info::clear()
{
    hipMemset(opt, 0x00, sizeof(float) * O);
    hipMemset(preact, 0x00, sizeof(float) * O);
}

void Layer_info::bp_clear()
{
    hipMemset(bp_opt, 0x00, sizeof(float) * O);
    hipMemset(bp_preact, 0x00, sizeof(float) * O);
    hipMemset(bp_weight, 0x00, sizeof(float) * M * N);
}

void Layer_info::save_model(FILE *model)
{
    char buffer[100000];
    fwrite((char*)&M, sizeof(int), 1, model);
    fwrite((char*)&N, sizeof(int), 1, model);

    hipMemcpy(buffer, (char*)bias, sizeof(float) * N, hipMemcpyDeviceToHost);
    fwrite(buffer, sizeof(float) * N, 1, model);
    hipMemcpy(buffer, (char*)weight, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    fwrite(buffer, sizeof(float) * M * N, 1, model);
}

void Layer_info::read_model(FILE *model)
{
    char buffer[100000];
    
    fread((char*)&M, sizeof(int), 1, model);
    fread((char*)&N, sizeof(int), 1, model);
    
    fread(buffer, sizeof(float) * N, 1, model);
    hipMemcpy(bias, (float*)buffer, sizeof(float) * N, hipMemcpyHostToDevice);
    fread(buffer, sizeof(float) * M * N, 1, model);
    hipMemcpy(weight, (float*)buffer, sizeof(float) * M * N, hipMemcpyHostToDevice);
}
